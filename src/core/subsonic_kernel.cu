#include "hip/hip_runtime.h"
#include "dns/dns.h"
#include "ui/progress_bar.h" // Include provided progress bar header
#include <hip/hip_runtime.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <netdb.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <linux/socket.h>
#include <errno.h>
#include <time.h>
#include <fcntl.h>
#include <netinet/ip.h>
#include <poll.h>

// Allowed characters for subdomains
__constant__ char d_allowed_chars[] = "abcdefghijklmnopqrstuvwxyz0123456789-";
const int num_chars = 37;
const int max_len = 5;
const int batch_size = 256; // Reduced to avoid rate limiting

// DNS resolvers
const char* resolvers[] = {"8.8.8.8", "1.1.1.1", "9.9.9.9", "8.8.4.4", "1.0.0.1", "1.0.0.2"};
const int num_resolvers = 6;
int active_resolvers[num_resolvers];
int num_active_resolvers = 0;

// Wildcard detection
char wildcard_response[512];
int wildcard_response_len = 0;
bool has_wildcard = false;

// Random delay range (milliseconds)
const int min_delay_ms = 200;
const int max_delay_ms = 1000;

// Buffer sizes
const int buffer_size = 262144;

// Global query ID counter
static uint16_t global_query_id = 0;

#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define SOCK_CHECK(err, msg) do { \
    if (err < 0) { \
        fprintf(stderr, "%s: %s\n", msg, strerror(errno)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// CUDA kernel to generate subdomains
__global__ void generate_subdomains(char* output_buffer, unsigned long long start_idx, int max_len, int num_chars) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x + start_idx;
    if (idx >= start_idx + batch_size) return;

    int len = 0;
    unsigned long long combo_idx = idx;
    unsigned long long total = 0;
    for (int i = 1; i <= max_len; ++i) {
        unsigned long long combos = 1ULL;
        for (int j = 0; j < i; ++j) combos *= num_chars;
        total += combos;
        if (idx < total) {
            len = i;
            combo_idx = idx - (total - combos);
            break;
        }
    }
    if (len == 0) return;

    char subdomain[64];
    for (int i = 0; i < len; ++i) {
        subdomain[i] = d_allowed_chars[combo_idx % num_chars];
        combo_idx /= num_chars;
    }
    subdomain[len] = '\0';

    if (subdomain[0] != '-' && (len == 1 || subdomain[len - 1] != '-')) {
        unsigned long long output_offset = (idx - start_idx) * 64;
        for (int i = 0; i <= len; ++i) {
            output_buffer[output_offset + i] = subdomain[i];
        }
    }
}

// Test resolver with blocking socket and poll
bool test_resolver(const char* ip, int sock) {
    struct sockaddr_in addr;
    memset(&addr, 0, sizeof(addr));
    addr.sin_family = AF_INET;
    addr.sin_port = htons(53);
    if (inet_pton(AF_INET, ip, &addr.sin_addr) != 1) {
        fprintf(stderr, "Invalid resolver IP: %s\n", ip);
        return false;
    }

    // Bind socket
    struct sockaddr_in local_addr;
    memset(&local_addr, 0, sizeof(local_addr));
    local_addr.sin_family = AF_INET;
    local_addr.sin_addr.s_addr = INADDR_ANY;
    local_addr.sin_port = 0;
    if (bind(sock, (struct sockaddr*)&local_addr, sizeof(local_addr)) < 0) {
        fprintf(stderr, "Bind failed for %s: %s\n", ip, strerror(errno));
        return false;
    }

    // Ensure blocking mode
    int flags = fcntl(sock, F_GETFL, 0);
    fcntl(sock, F_SETFL, flags & ~O_NONBLOCK);

    char query[512];
    int query_len = create_dns_query(query, "example", "com", 0x1234, 1);
    if (query_len == 0) {
        fprintf(stderr, "Failed to create DNS query for %s\n", ip);
        return false;
    }

#ifndef NDEBUG
    // Log raw query
    printf("Raw query for %s: ", ip);
    for (int i = 0; i < query_len; ++i) {
        printf("%02x ", (unsigned char)query[i]);
    }
    printf("\n");
#endif

    const int max_retries = 3;
    for (int retry = 0; retry < max_retries; ++retry) {
        int sent = sendto(sock, query, query_len, 0, (struct sockaddr*)&addr, sizeof(addr));
        if (sent < 0) {
            fprintf(stderr, "sendto failed for %s (retry %d): %s\n", ip, retry, strerror(errno));
            continue;
        }

        struct pollfd pfd = {sock, POLLIN, 0};
        int timeout_ms = 10000; // 10s
        int polled = poll(&pfd, 1, timeout_ms);
        if (polled < 0) {
            fprintf(stderr, "poll failed for %s (retry %d): %s\n", ip, retry, strerror(errno));
            continue;
        }
        if (polled == 0) {
            fprintf(stderr, "poll timeout for %s (retry %d)\n", ip, retry);
            continue;
        }

        char response[512];
        int recvd = recvfrom(sock, response, sizeof(response), 0, NULL, NULL);
        if (recvd > 0) {
            char record_data[256];
            int record_len;
            int result = parse_dns_response(response, recvd, record_data, &record_len);
            if (result == 1) {
#ifndef NDEBUG
                printf("Successfully parsed DNS response for %s: %d bytes\n", ip, record_len);
#endif
                return true;
            } else if (result == -1) {
                fprintf(stderr, "NXDOMAIN for %s (retry %d)\n", ip, retry);
            } else {
                fprintf(stderr, "Failed to parse DNS response for %s, received %d bytes\n", ip, recvd);
#ifndef NDEBUG
                fprintf(stderr, "Raw response: ");
                for (int i = 0; i < recvd; ++i) fprintf(stderr, "%02x ", (unsigned char)response[i]);
                fprintf(stderr, "\n");
#endif
            }
        } else {
            fprintf(stderr, "recvfrom failed for %s (retry %d): %s\n", ip, retry, strerror(errno));
        }
        usleep(500000); // 500ms delay between retries
    }
    return false;
}

// Detect wildcard
void detect_wildcard(const char* target, int* socks, struct sockaddr_in* dns_servers, int num_servers) {
    srand(time(NULL));
    char random_subdomain[20];
    const char* allowed_chars = "abcdefghijklmnopqrstuvwxyz0123456789-";
    int attempts = 0;
    const int max_attempts = 10;

    while (attempts < max_attempts) {
        // Generate random subdomain
        for (int j = 0; j < 15; ++j) {
            random_subdomain[j] = allowed_chars[rand() % num_chars];
        }
        random_subdomain[15] = '\0';
#ifndef NDEBUG
        printf("Testing wildcard: %s.%s\n", random_subdomain, target);
#endif

        char query[512];
        int query_len = create_dns_query(query, random_subdomain, target, global_query_id++, 1);
        if (query_len == 0) {
            fprintf(stderr, "Failed to create wildcard query for %s\n", random_subdomain);
            attempts++;
            continue;
        }

        struct msghdr msg = {0};
        struct iovec iov = {query, (size_t)query_len};
        msg.msg_name = &dns_servers[attempts % num_servers];
        msg.msg_namelen = sizeof(struct sockaddr_in);
        msg.msg_iov = &iov;
        msg.msg_iovlen = 1;

        int sent = sendmsg(socks[attempts % num_servers], &msg, 0);
        if (sent < 0) {
            fprintf(stderr, "Wildcard sendmsg failed for %s: %s\n", random_subdomain, strerror(errno));
            attempts++;
            continue;
        }

        char response[512];
        struct iovec response_iov = {response, sizeof(response)};
        struct msghdr response_msg = {0};
        response_msg.msg_name = &dns_servers[attempts % num_servers];
        response_msg.msg_namelen = sizeof(struct sockaddr_in);
        response_msg.msg_iov = &response_iov;
        response_msg.msg_iovlen = 1;

        // Use poll for blocking receive
        struct pollfd pfd = {socks[attempts % num_servers], POLLIN, 0};
        int timeout_ms = 2000; // 2s
        int polled = poll(&pfd, 1, timeout_ms);
        if (polled <= 0) {
            fprintf(stderr, "Wildcard poll %s for %s.%s\n", polled == 0 ? "timeout" : "failed", random_subdomain, target);
            attempts++;
            continue;
        }

        int recvd = recvmsg(socks[attempts % num_servers], &response_msg, 0);
        if (recvd > 0) {
            char record_data[256];
            int record_len;
            int result = parse_dns_response(response, recvd, record_data, &record_len);
            if (result == 1) {
                memcpy(wildcard_response, record_data, record_len);
                wildcard_response_len = record_len;
                has_wildcard = true;
                printf("Wildcard detected for *.%s with IP: %d.%d.%d.%d\n", target,
                       (unsigned char)record_data[0], (unsigned char)record_data[1],
                       (unsigned char)record_data[2], (unsigned char)record_data[3]);
                return;
            } else if (result == -1) {
                // NXDOMAIN is expected for non-wildcard domains
                attempts++;
            } else {
                fprintf(stderr, "Failed to parse wildcard response for %s.%s, received %d bytes\n", random_subdomain, target, recvd);
            }
        } else {
            fprintf(stderr, "Wildcard recvmsg failed for %s.%s: %s\n", random_subdomain, target, strerror(errno));
            attempts++;
        }
    }

    printf("No wildcard detected for *.%s after %d attempts\n", target, max_attempts);
}

// Match response ID to query ID
bool check_response_id(const char* response, int recvd, uint16_t query_id) {
    if (recvd < sizeof(dns_header_t)) return false;
    dns_header_t* header = (dns_header_t*)response;
    bool match = ntohs(header->id) == query_id;
#ifndef NDEBUG
    if (!match) {
        printf("ID mismatch: expected %u, got %u\n", query_id, ntohs(header->id));
    }
#endif
    return match;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <target_hostname>\n", argv[0]);
        return 1;
    }
    const char* target = argv[1];

    // Set process priority
    errno = 0;
    int nice_val = nice(-5);
    if (nice_val == -1 && errno != 0) {
        fprintf(stderr, "Warning: Failed to set nice value to -5 (%s)\n", strerror(errno));
    } else {
#ifndef NDEBUG
        printf("Set process nice value to %d\n", nice_val);
#endif
    }

    // Seed random number generator
    srand(time(NULL));

    // Initialize sockets
    int socks[num_resolvers];
    for (int i = 0; i < num_resolvers; ++i) {
        socks[i] = socket(AF_INET, SOCK_DGRAM, 0);
        SOCK_CHECK(socks[i], "socket creation failed");
        int opt = 1;
        setsockopt(socks[i], SOL_SOCKET, SO_REUSEADDR, &opt, sizeof(opt));
        int mtu_opt = IP_PMTUDISC_DO;
        setsockopt(socks[i], IPPROTO_IP, IP_MTU_DISCOVER, &mtu_opt, sizeof(mtu_opt));
        setsockopt(socks[i], SOL_SOCKET, SO_SNDBUF, &buffer_size, sizeof(buffer_size));
        setsockopt(socks[i], SOL_SOCKET, SO_RCVBUF, &buffer_size, sizeof(buffer_size));
        // Ensure blocking mode
        int flags = fcntl(socks[i], F_GETFL, 0);
        fcntl(socks[i], F_SETFL, flags & ~O_NONBLOCK);
    }

    // Test resolvers
    for (int i = 0; i < num_resolvers; ++i) {
        if (test_resolver(resolvers[i], socks[i])) {
            active_resolvers[num_active_resolvers++] = i;
#ifndef NDEBUG
            printf("Resolver %s is active\n", resolvers[i]);
#endif
        } else {
#ifndef NDEBUG
            printf("Resolver %s is inactive\n", resolvers[i]);
#endif
            close(socks[i]);
            socks[i] = -1;
        }
    }
    if (num_active_resolvers == 0) {
        fprintf(stderr, "No active resolvers found\n");
        for (int i = 0; i < num_resolvers; ++i) {
            if (socks[i] >= 0) close(socks[i]);
        }
        return 1;
    }

    // Set up DNS server addresses
    struct sockaddr_in dns_servers[num_active_resolvers];
    for (int i = 0; i < num_active_resolvers; ++i) {
        memset(&dns_servers[i], 0, sizeof(dns_servers[i]));
        dns_servers[i].sin_family = AF_INET;
        dns_servers[i].sin_port = htons(53);
        inet_pton(AF_INET, resolvers[active_resolvers[i]], &dns_servers[i].sin_addr);
    }

    // Detect wildcard
    detect_wildcard(target, socks, dns_servers, num_active_resolvers);

    // Open output file
    FILE* output_file = fopen("valid_subdomains.txt", "w");
    if (!output_file) {
        fprintf(stderr, "Failed to open output file: %s\n", strerror(errno));
        for (int i = 0; i < num_resolvers; ++i) {
            if (socks[i] >= 0) close(socks[i]);
        }
        return 1;
    }

    // CUDA setup
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    char* h_output_buffer = (char*)malloc(batch_size * 64 * sizeof(char));
    char* d_output_buffer;
    CUDA_CHECK(hipMalloc(&d_output_buffer, batch_size * 64 * sizeof(char)));

    // DNS query buffers
    char query_buffers[batch_size][512];
    struct mmsghdr send_msgs[batch_size];
    struct mmsghdr recv_msgs[batch_size];
    struct iovec send_iovecs[batch_size];
    struct iovec recv_iovecs[batch_size];
    struct sockaddr_in addrs[batch_size];
    char response_buffers[batch_size][512];
    uint16_t query_ids[batch_size]; // Store query IDs
    int query_sockets[batch_size];   // Store socket index for each query
    char subdomains[batch_size][64]; // Store subdomains for matching
    int id_mismatches = 0;           // Track mismatches per batch
    int timeouts = 0;                // Track timeouts per batch

    // Main loop
    unsigned long long start_idx = 0;
    unsigned long long max_combinations = 0;
    for (int i = 1; i <= max_len; ++i) {
        max_combinations += (i == 1 ? 37 : 36 * 36 * (unsigned long long)pow(37, i-2));
    }
#ifndef NDEBUG
    printf("Total combinations to test: %llu\n", max_combinations);
#endif

    // Initialize progress bar
    char progress_message[128];
    snprintf(progress_message, sizeof(progress_message), "Querying %s.%s", "", target);
    progress_bar(progress_message, start_idx, max_combinations);

    while (start_idx < max_combinations) {
        // Update progress bar
        snprintf(progress_message, sizeof(progress_message), "Querying %s.%s", subdomains[0][0] ? subdomains[0] : "", target);
        progress_bar(progress_message, start_idx, max_combinations);

        // Generate subdomains
        int threads_per_block = 256;
        int blocks = (batch_size + threads_per_block - 1) / threads_per_block;
        generate_subdomains<<<blocks, threads_per_block, 0, stream>>>(d_output_buffer, start_idx, max_len, num_chars);
        CUDA_CHECK(hipStreamSynchronize(stream));
        CUDA_CHECK(hipMemcpyAsync(h_output_buffer, d_output_buffer, batch_size * 64 * sizeof(char), hipMemcpyDeviceToHost, stream));
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Log generated subdomains
        int valid_subdomains = 0;
        for (int i = 0; i < batch_size; ++i) {
            char* subdomain = &h_output_buffer[i * 64];
            if (subdomain[0] != '\0' && strcmp(subdomain, "") != 0) {
#ifndef NDEBUG
                if (valid_subdomains < 5) { // Limit logging to first 5 for brevity
                    printf("Generated subdomain: %s.%s\n", subdomain, target);
                }
#endif
                valid_subdomains++;
            }
        }
#ifndef NDEBUG
        printf("Total valid subdomains in batch: %d\n", valid_subdomains);
#endif

        // Prepare DNS queries
        int valid_queries = 0;
        id_mismatches = 0;
        timeouts = 0;
        for (int i = 0; i < batch_size; ++i) {
            char* subdomain = &h_output_buffer[i * 64];
            if (subdomain[0] == '\0' || strcmp(subdomain, "") == 0) continue; // Skip empty subdomains

            query_ids[valid_queries] = global_query_id++; // Unique query ID
            query_sockets[valid_queries] = active_resolvers[valid_queries % num_active_resolvers]; // Assign resolver
            strncpy(subdomains[valid_queries], subdomain, 64); // Store subdomain
            int query_len = create_dns_query(query_buffers[valid_queries], subdomain, target, query_ids[valid_queries], 1);
            if (query_len == 0) continue;
            send_iovecs[valid_queries].iov_base = query_buffers[valid_queries];
            send_iovecs[valid_queries].iov_len = query_len;
            send_msgs[valid_queries].msg_hdr.msg_iov = &send_iovecs[valid_queries];
            send_msgs[valid_queries].msg_hdr.msg_iovlen = 1;
            send_msgs[valid_queries].msg_hdr.msg_name = &addrs[valid_queries];
            send_msgs[valid_queries].msg_hdr.msg_namelen = sizeof(addrs[valid_queries]);
            memcpy(&addrs[valid_queries], &dns_servers[valid_queries % num_active_resolvers], sizeof(struct sockaddr_in));
            // Prepare receive message
            recv_iovecs[valid_queries].iov_base = response_buffers[valid_queries];
            recv_iovecs[valid_queries].iov_len = sizeof(response_buffers[valid_queries]);
            recv_msgs[valid_queries].msg_hdr.msg_iov = &recv_iovecs[valid_queries];
            recv_msgs[valid_queries].msg_hdr.msg_iovlen = 1;
            recv_msgs[valid_queries].msg_hdr.msg_name = &addrs[valid_queries];
            recv_msgs[valid_queries].msg_hdr.msg_namelen = sizeof(addrs[valid_queries]);
            recv_msgs[valid_queries].msg_hdr.msg_control = NULL;
            recv_msgs[valid_queries].msg_hdr.msg_controllen = 0;
            valid_queries++;
        }
#ifndef NDEBUG
        printf("Prepared %d valid queries\n", valid_queries);
#endif

        // Send and receive queries synchronously
        int sent_queries = 0;
        int received_responses = 0;
        if (valid_queries > 0) {
            // Send queries
            for (int i = 0; i < valid_queries; i += num_active_resolvers) {
                int batch_size = (valid_queries - i < num_active_resolvers) ? valid_queries - i : num_active_resolvers;
                for (int j = 0; j < batch_size; ++j) {
                    int idx = i + j;
                    int sock_idx = query_sockets[idx];
                    int sent = sendmsg(socks[sock_idx], &send_msgs[idx].msg_hdr, 0);
                    if (sent < 0) {
                        fprintf(stderr, "sendmsg failed for query %d (%s.%s): %s\n",
                                idx, subdomains[idx], target, strerror(errno));
                        continue;
                    }
                    sent_queries++;
                }

                // Receive responses
                for (int attempt = 0; attempt < 3 && received_responses < sent_queries; ++attempt) {
                    for (int j = 0; j < batch_size; ++j) {
                        int idx = i + j;
                        int sock_idx = query_sockets[idx];
                        struct pollfd pfd = {socks[sock_idx], POLLIN, 0};
                        int timeout_ms = 15000; // 15s timeout
                        int polled = poll(&pfd, 1, timeout_ms);
                        if (polled <= 0) {
                            timeouts++;
#ifndef NDEBUG
                            printf("Poll %s for query %d (%s.%s)\n",
                                   polled == 0 ? "timeout" : "failed", idx, subdomains[idx], target);
#endif
                            continue;
                        }

                        int recvd = recvmsg(socks[sock_idx], &recv_msgs[idx].msg_hdr, 0);
                        if (recvd > 0) {
                            if (!check_response_id(response_buffers[idx], recvd, query_ids[idx])) {
                                id_mismatches++;
#ifndef NDEBUG
                                printf("Skipped response for %s.%s: ID mismatch\n", subdomains[idx], target);
#endif
                                continue;
                            }
                            char record_data[256];
                            int record_len;
                            int result = parse_dns_response(response_buffers[idx], recvd, record_data, &record_len);
                            if (result == 1) {
                                if (!has_wildcard || memcmp(record_data, wildcard_response, record_len) != 0) {
                                    if (subdomains[idx][0] != '\0') { // Ensure valid subdomain
                                        fprintf(output_file, "Valid subdomain: %s.%s\n", subdomains[idx], target);
                                        printf("Valid subdomain: %s.%s\n", subdomains[idx], target); // Removed leading \n
                                        fflush(stdout); // Ensure immediate display
                                    }
                                }
                                received_responses++;
                            } else if (result == -1) {
                                // NXDOMAIN or no A records: silently skip
                                received_responses++;
                            } else {
                                fprintf(stderr, "Malformed response for query %d (%s.%s), received %d bytes\n",
                                        idx, subdomains[idx], target, recvd);
                            }
                        } else {
                            fprintf(stderr, "recvmsg failed for query %d (%s.%s): %s\n",
                                    idx, subdomains[idx], target, strerror(errno));
                        }
                    }
                    if (received_responses < sent_queries && attempt < 2) {
#ifndef NDEBUG
                        printf("Retrying receive, attempt %d, %d responses of %d sent queries\n",
                               attempt + 1, received_responses, sent_queries);
#endif
                        usleep(500000); // 500ms delay before retry
                    }
                }
            }
        }
        printf("Sent %d queries, received %d responses, %d timeouts, %d ID mismatches\n",
               sent_queries, received_responses, timeouts, id_mismatches);
        fflush(stdout); // Ensure query stats are displayed immediately

        // Random delay
        int delay_ms = min_delay_ms + (rand() % (max_delay_ms - min_delay_ms + 1));
        struct timespec delay = {0, delay_ms * 1000000};
        nanosleep(&delay, NULL);

        start_idx += batch_size;
    }

    // Final progress bar update
    snprintf(progress_message, sizeof(progress_message), "Completed");
    progress_bar(progress_message, max_combinations, max_combinations);

    // Cleanup
    free(h_output_buffer);
    CUDA_CHECK(hipFree(d_output_buffer));
    CUDA_CHECK(hipStreamDestroy(stream));
    for (int i = 0; i < num_resolvers; ++i) {
        if (socks[i] >= 0) close(socks[i]);
    }
    fclose(output_file);
    return 0;
}
